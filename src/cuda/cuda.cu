#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

const double _INFINITY = DBL_MAX;
const int _MAX_DISTANCE = 5;
const int BLOCK_SIZE = 64;

void cudaCheckError() {
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) { \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
}

void printDeviceInfo() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("CUDA Device: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max threads dim: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max grid size: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

__device__ double atomicMinDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(min(val, __longlong_as_double(assumed))));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void cosine_similarity_kernel(double *graph, double *D, int n) {
    const double _INFINITY = DBL_MAX;
    int i = blockIdx.x;
    int j = threadIdx.x + blockIdx.y * blockDim.x;

    if (j <= i || j >= n)
        return;

    double dot = 0.0, norm_i = 0.0, norm_j = 0.0;

    for (int k = 0; k < n; k++) {
        dot += graph[i * n + k] * graph[j * n + k];
        norm_i += graph[i * n + k] * graph[i * n + k];
        norm_j += graph[j * n + k] * graph[j * n + k];
    }

    norm_i = sqrt(norm_i);
    norm_j = sqrt(norm_j);

    double similarity;
    if (norm_i == 0 || norm_j == 0)
        similarity = 0;
    else
        similarity = dot / (norm_i * norm_j);

    double inverse_similarity = (similarity == 0) ? _INFINITY : 1 - similarity;

    D[i * n + j] = inverse_similarity;
    D[j * n + i] = inverse_similarity;
}

__global__ void fw_kernel(double *D, int n, int k, double r) {
    int i = blockIdx.x;
    int j = threadIdx.x + blockIdx.y * blockDim.x;

    if (i >= n || j >= n || i == j)
        return;

    double a = D[i * n + k];
    double b = D[k * n + j];

    if (a == DBL_MAX || b == DBL_MAX)
        return;

    double t;
    if (r == 1.0) {
        t = a + b;
    } else if (r == 2.0) {
        t = sqrt(a*a + b*b);
    } else if (r == DBL_MAX) {
        t = fmax(a, b);
    } else {
        t = pow((pow(a, r) + pow(b, r)), (1.0 / r));
    }

    if (t < D[i * n + j]) {
        atomicMinDouble(&D[i * n + j], t);
    }
}

void floyd_warshall_cuda(double *D, int n, double r) {
    double *d_D;
    size_t size = n * n * sizeof(double);

    hipMalloc(&d_D, size);
    hipMemcpy(d_D, D, size, hipMemcpyHostToDevice);
    cudaCheckError();

    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim(n, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);

    for (int k = 0; k < n; k++) {
        fw_kernel<<<gridDim, blockDim>>>(d_D, n, k, r);
        hipDeviceSynchronize();
        cudaCheckError();
    }

    hipMemcpy(D, d_D, size, hipMemcpyDeviceToHost);
    cudaCheckError();

    hipFree(d_D);
    cudaCheckError();
}

int find_index(char **array, int size, const char *word) {
    for (int i = 0; i < size; i++) {
        if (strcmp(array[i], word) == 0) {
            return i;
        }
    }
    return -1;
}

int word_exists(char **array, int size, const char *word) {
    return find_index(array, size, word) != -1;
}

int add_word(char ***array, int *size, const char *word) {
    if (!word_exists(*array, *size, word)) {
        *array = (char **)realloc(*array, (*size + 1) * sizeof(char *));
        (*array)[*size] = strdup(word);
        (*size)++;
        return 1;
    }
    return 0;
}

int compare_strings(const void *a, const void *b) {
    return strcmp(*(const char **)a, *(const char **)b);
}

int main() {
    printf("===============================================\n");
    printf("PATHFINDER NETWORK (CUDA Naive Implementation)\n");
    printf("===============================================\n");

    printDeviceInfo();

    char buffer[1024];
    char **text = NULL;
    int text_size = 0;

    while (scanf("%s", buffer) == 1) {
        text = (char **)realloc(text, (text_size + 1) * sizeof(char *));
        text[text_size] = strdup(buffer);
        text_size++;
    }

    printf("Text size:\t%d\n", text_size);

    clock_t start = clock();

    char **wordSet = NULL;
    int wordSetSize = 0;

    for (int i = 0; i < text_size; i++) {
        add_word(&wordSet, &wordSetSize, text[i]);
    }

    qsort(wordSet, wordSetSize, sizeof(char *), compare_strings);

    printf("Unique words:\t%d\n", wordSetSize);

    clock_t wordSetEnd = clock();
    printf("Word Set:\t%ld ms\n", (wordSetEnd - start) * 1000 / CLOCKS_PER_SEC);

    int n = wordSetSize;

    double *graph = (double *)calloc(n * n, sizeof(double));
    for (int i = 0; i < text_size; i++) {
        int token_i = find_index(wordSet, wordSetSize, text[i]);
        int max_neighbor = (i + 1 + _MAX_DISTANCE < text_size) ? i + 1 + _MAX_DISTANCE : text_size;
        for (int j = i + 1; j < max_neighbor; j++) {
            int token_j = find_index(wordSet, wordSetSize, text[j]);
            if (token_i != token_j) {
                graph[token_i * n + token_j]++;
                graph[token_j * n + token_i]++;
            }
        }
    }

    clock_t graphInitEnd = clock();
    printf("Graph Init:\t%ld ms\n", (graphInitEnd - wordSetEnd) * 1000 / CLOCKS_PER_SEC);

    double *D = (double *)malloc(n * n * sizeof(double));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) {
                D[i * n + j] = 0;
            } else {
                D[i * n + j] = _INFINITY;
            }
        }
    }

    double *d_graph, *d_D;
    size_t size = n * n * sizeof(double);

    hipMalloc(&d_graph, size);
    hipMalloc(&d_D, size);

    hipMemcpy(d_graph, graph, size, hipMemcpyHostToDevice);
    hipMemcpy(d_D, D, size, hipMemcpyHostToDevice);
    cudaCheckError();

    dim3 gridDim(n, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockDim(BLOCK_SIZE);

    cosine_similarity_kernel<<<gridDim, blockDim>>>(d_graph, d_D, n);
    hipDeviceSynchronize();
    cudaCheckError();

    hipMemcpy(D, d_D, size, hipMemcpyDeviceToHost);
    hipFree(d_graph);
    hipFree(d_D);
    cudaCheckError();

    clock_t similarityEnd = clock();
    printf("Similarity:\t%ld ms\n", (similarityEnd - graphInitEnd) * 1000 / CLOCKS_PER_SEC);

    // const double r = 1;
    // const double r = 2;
    const double r = _INFINITY;

    floyd_warshall_cuda(D, n, r);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i != j && graph[i * n + j] > 0 && graph[i * n + j] < D[i * n + j]) {
                D[i * n + j] = graph[i * n + j];
            }
        }
    }

    clock_t pfEnd = clock();
    printf("Pathfinder:\t%ld ms\n", (pfEnd - similarityEnd) * 1000 / CLOCKS_PER_SEC);
    printf("Total:\t%ld ms\n", (pfEnd - start) * 1000 / CLOCKS_PER_SEC);
    printf("===============================================\n");
    printf("RESULT\n");
    printf("===============================================\n");

    for (int i = 0; i < n; i++) {
        for (int j = i + 1; j < n; j++) {
            if (D[i * n + j] == _INFINITY) {
                printf("%s %s inf\n", wordSet[i], wordSet[j]);
            } else {
                printf("%s %s %f\n", wordSet[i], wordSet[j], D[i * n + j]);
            }
        }
    }

    for (int i = 0; i < text_size; i++) {
        free(text[i]);
    }
    free(text);

    for (int i = 0; i < wordSetSize; i++) {
        free(wordSet[i]);
    }
    free(wordSet);

    free(graph);
    free(D);

    return 0;
}