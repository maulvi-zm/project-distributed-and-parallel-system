#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

const double _INFINITY = DBL_MAX;
const int _MAX_DISTANCE = 5;

void update_row(double **D, const int i, const int n, const int k,
                const double r) {
  for (int j = 0; j < n; j++) {
    if (i == j)
      continue;

    double a = D[i][k];
    double b = D[k][j];
    double t = pow((pow(a, r) + pow(b, r)), (1.0 / r));

    if (t < D[i][j]) {
      D[i][j] = t;
    }
  }
}

__global__ void update_row_kernel(double *D, int n, int i, int k, double r){
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (j < n && i != j){
    double a = D[i * n + k];
    double b = D[k * n + j];
    double t = pow((pow(a, r) + pow(b, r)), (1.0 / r));
    if (t < D[i * n + j]) {
      D[i * n + j] = t;
    }
  }
}

void floyd_warshall(double **D, int q, int r){
  int n = q + 1;
  double *d_D;
  hipMalloc((void **)&d_D, n * n * sizeof(double));

  double *D_flat = (double *) malloc (n * n * sizeof(double));

  for (int i = 0; i < n; i++){
    for (int j = 0; j < n; j++) {
      D_flat[i * n + j] = D[i][j];
    }
  }

  for (int i = 0; i < n; i++){ // isunya sama kek mpi, klo ga di-copy per row, ngecrash
    hipMemcpy(d_D + i * n, D_flat + i * n, n * sizeof(double), hipMemcpyHostToDevice);
  }

  for (int k = 0; k < n; k++) {
    for (int i = 0; i < n; i++) {
      int blockSize = 256;
      int numBlocks = (n + blockSize - 1) / blockSize;
      update_row_kernel<<<numBlocks, blockSize>>>(d_D, n, i, k, r);
    }
    hipDeviceSynchronize();
  }

  
  for (int i = 0; i < n; i++){
    hipMemcpy(D[i], d_D + i * n, n * sizeof(double), hipMemcpyDeviceToHost);
  }
  hipFree(d_D);
}

double **pathfinder_network(double **graph, int n, int q, int r) {
  double **D = (double **)malloc(n * sizeof(double *));
  for (int i = 0; i < n; i++) {
    D[i] = (double *)malloc(n * sizeof(double));
    for (int j = 0; j < n; j++) {
      D[i][j] = graph[i][j];
    }
  }

  floyd_warshall(D, q, r);

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (graph[i][j] < D[i][j]) {
        D[i][j] = graph[i][j];
      }
    }
  }

  return D;
}

double cosine_similarity(double *a, double *b, int n) {
  double dot = 0.0, norm_a = 0.0, norm_b = 0.0;

  for (int i = 0; i < n; i++) {
    dot += a[i] * b[i];
    norm_a += a[i] * a[i];
    norm_b += b[i] * b[i];
  }

  norm_a = sqrt(norm_a);
  norm_b = sqrt(norm_b);

  if (norm_a == 0 || norm_b == 0)
    return 0;

  return dot / (norm_a * norm_b);
}

int find_index(char **array, int size, const char *word) {
  for (int i = 0; i < size; i++) {
    if (strcmp(array[i], word) == 0) {
      return i;
    }
  }
  return -1;
}

int word_exists(char **array, int size, const char *word) {
  return find_index(array, size, word) != -1;
}

int add_word(char ***array, int *size, const char *word) {
  if (!word_exists(*array, *size, word)) {
    *array = (char**) realloc(*array, (*size + 1) * sizeof(char *));
    (*array)[*size] = strdup(word);
    (*size)++;
    return 1;
  }
  return 0;
}

int compare_strings(const void *a, const void *b) {
  return strcmp(*(const char **)a, *(const char **)b);
}

int main() {
  printf("===============================================\n");
  printf("PATHFINDER NETWORK\n");
  printf("===============================================\n");

  char buffer[1024];
  char **text = NULL;
  int text_size = 0;

  while (scanf("%s", buffer) == 1) {
    text = (char**) realloc(text, (text_size + 1) * sizeof(char *));
    text[text_size] = strdup(buffer);
    text_size++;
  }

  printf("Text size:\t%d\n", text_size);

  clock_t start = clock();

  char **wordSet = NULL;
  int wordSetSize = 0;

  for (int i = 0; i < text_size; i++) {
    add_word(&wordSet, &wordSetSize, text[i]);
  }

  qsort(wordSet, wordSetSize, sizeof(char *), compare_strings);

  printf("Unique words:\t%d\n", wordSetSize);

  clock_t wordSetEnd = clock();
  printf("Word Set:\t%ld s\n", (wordSetEnd - start) / CLOCKS_PER_SEC);

  int n = wordSetSize;

  double **graph = (double **)malloc(n * sizeof(double *));
  for (int i = 0; i < n; i++) {
    graph[i] = (double *)calloc(n, sizeof(double));
  }

  for (int i = 0; i < text_size; i++) {
    int token_i = find_index(wordSet, wordSetSize, text[i]);

    int max_neighbor =
        (i + 1 + _MAX_DISTANCE < text_size) ? i + 1 + _MAX_DISTANCE : text_size;
    for (int j = i + 1; j < max_neighbor; j++) {
      int token_j = find_index(wordSet, wordSetSize, text[j]);
      if (token_i != token_j) {
        graph[token_i][token_j]++;
        graph[token_j][token_i]++;
      }
    }
  }

  clock_t graphInitEnd = clock();
  printf("Graph Init:\t%ld s\n", (graphInitEnd - wordSetEnd) / CLOCKS_PER_SEC);

  double **D = (double **)malloc(n * sizeof(double *));
  for (int i = 0; i < n; i++) {
    D[i] = (double *)malloc(n * sizeof(double));
  }

  for (int i = 0; i < n; i++) {
    D[i][i] = 0;
    for (int j = i + 1; j < n; j++) {
      double similarity = cosine_similarity(graph[i], graph[j], n);
      double inverse_similarity;
      if (similarity == 0) {
        inverse_similarity = _INFINITY;
      } else {
        inverse_similarity = 1 - similarity;
      }
      D[i][j] = inverse_similarity;
      D[j][i] = inverse_similarity;
    }
  }

  clock_t similarityEnd = clock();
  printf("Similarity:\t%ld s\n",
         (similarityEnd - graphInitEnd) / CLOCKS_PER_SEC);

  const int q = n - 1;
  const double r = 1;

  double **pf_net = pathfinder_network(D, n, q, r);

  clock_t pfEnd = clock();
  printf("Pathfinder:\t%ld s\n", (pfEnd - similarityEnd) / CLOCKS_PER_SEC);
  printf("Total:\t%ld s\n", (pfEnd - start) / CLOCKS_PER_SEC);
  printf("===============================================\n");
  printf("RESULT\n");
  printf("===============================================\n");

  for (int i = 0; i < n; i++) {
    for (int j = i + 1; j < n; j++) {
      if (pf_net[i][j] == _INFINITY) {
        printf("%s %s inf\n", wordSet[i], wordSet[j]);
      } else {
        printf("%s %s %f\n", wordSet[i], wordSet[j], pf_net[i][j]);
      }
    }
  }

  for (int i = 0; i < text_size; i++) {
    free(text[i]);
  }
  free(text);

  for (int i = 0; i < wordSetSize; i++) {
    free(wordSet[i]);
  }
  free(wordSet);

  for (int i = 0; i < n; i++) {
    free(graph[i]);
    free(D[i]);
    free(pf_net[i]);
  }
  free(graph);
  free(D);
  free(pf_net);

  return 0;
}
